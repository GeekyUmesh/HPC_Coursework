#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
  
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library 
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
    nvcc -o Password_Crack_Cuda Password_Crack_Cuda.cu 
    ./Password_Crack_Cuda
   
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

int time_difference(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

__device__ int is_a_match(char *attempt) {
  char plain_password1[] = "AS1234";
  char plain_password2[] = "UM9849";
  char plain_password3[] = "SN7832";
  char plain_password4[] = "RP8720";


  char *first = attempt;
  char *second = attempt;
  char *third = attempt;
  char *fourth = attempt;
  char *password1 = plain_password1;
  char *password2 = plain_password2;
  char *password3 = plain_password3;
  char *password4 = plain_password4;

  while(*first == *password1) { 
   if(*first == '\0') 
    {
	printf("Password Found: %s\n",plain_password1);
      break;
    }

    first++;
    password1++;
  }
	
  while(*second == *password2) { 
   if(*second == '\0') 
    {
	printf("Password Found: %s\n",plain_password2);
      break;
    }

    second++;
    password2++;
  }

  while(*third == *password3) { 
   if(*third == '\0') 
    {
	printf("Password Found: %s\n",plain_password3);
      break;
    }

    third++;
    password3++;
  }

  while(*fourth == *password4) { 
   if(*fourth == '\0') 
    {
	printf("Password Found: %s\n",plain_password4);
      return 1;
    }

    fourth++;
    password4++;
  }
  return 0;

}


/****************************************************************************
  The kernel function assume that there will be only one thread and uses 
  nested loops to generate all possible passwords and test whether they match
  the hidden password.
*****************************************************************************/

__global__ void  kernel() {
char w, x, y, z;
  
  char password[7];
  password[6] = '\0';

int q = blockIdx.x+65;
int t = threadIdx.x+65;
char firstInitial = q; 
char secondInitial = t; 
    
password[0] = firstInitial;
password[1] = secondInitial;
	for(w='0'; w<='9'; w++){
	  for(x='0'; x<='9'; x++){
	   for(y='0'; y<='9'; y++){
	     for(z='0'; z<='9'; z++){
	        password[2] = w;
	        password[3] = x;
	        password[4] = y;
	        password[5] = z; 
	      if(is_a_match(password)) {
		//printf("Success");
	      } 
             else {
	       // printf("tried: %s\n", password);		  
	         }
	      }
	   }
	}
    }
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

  kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed
					, (time_elapsed/1.0e9)); 
  return 0;
}



